
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMul(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int *a, *b, *c; // host matrices
    int *d_a, *d_b, *d_c; // device matrices
    int N; // Size of matrices

    // Input matrix size from user
    printf("Enter the size of matrices: ");
    scanf("%d", &N);

    int size = N * N * sizeof(int);

    // Allocate memory for host matrices
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize host matrices
    printf("Enter elements for Matrix A:\n");
    for (int i = 0; i < N * N; i++) {
        scanf("%d", &a[i]);
    }

    printf("Enter elements for Matrix B:\n");
    for (int i = 0; i < N * N; i++) {
        scanf("%d", &b[i]);
    }

    // Allocate memory for device matrices
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy host matrices to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Result Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", c[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
