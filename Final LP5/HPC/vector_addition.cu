
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int *a, *b, *c; // host vectors
    int *d_a, *d_b, *d_c; // device vectors
    int N; // Size of vectors

    // Input size of vectors from user
    printf("Enter the size of vectors: ");
    scanf("%d", &N);

    int size = N * sizeof(int);

    // Allocate memory for host vectors
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize host vectors
    printf("Enter elements for Vector A:\n");
    for (int i = 0; i < N; i++) {
        scanf("%d", &a[i]);
    }

    printf("Enter elements for Vector B:\n");
    for (int i = 0; i < N; i++) {
        scanf("%d", &b[i]);
    }

    // Allocate memory for device vectors
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy host vectors to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result vector
    printf("Result Vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
